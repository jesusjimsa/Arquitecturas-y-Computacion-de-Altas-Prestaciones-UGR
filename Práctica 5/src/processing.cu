#include "hip/hip_runtime.h"
#include <processing.h>
#include <stdio.h>

using namespace std;

__global__
void gaussianKernel(int *original, int *original_width, int *original_height, int *size, int *imgblur){
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	printf("jeje\n");

	// Declaraciones
	unsigned int blurpixel;
	signed int dx, dy;
	unsigned int pixelweight;
	unsigned int pixel;

	// Declarar el array de pesos para el difuminado gaussiano
	int weighting[5][5] = {
		{2, 4, 5, 4, 2},
		{4, 9, 12, 9, 4},
		{5, 12, 15, 12, 5},
		{4, 9, 12, 9, 4},
		{2, 4, 5, 4, 2}
	};

	if(id < (*size)){
		// Aplicar el flitro a cada pixel
		for (int x = 2; x <= (*original_width) - 2; x++){
			for (int y = 2; y <= (*original_height) - 2; y++){

				// Limpiar blurpixel
				blurpixel = 0;

				// +-2 para cada pixel y calcular el peso
				for (dx = -2; dx <= 2; dx++){
					for (dy = -2; dy <= 2; dy++){
						pixelweight = weighting[dx + 2][dy + 2];


						// Conseguir pixel
						if(x + dx >= (*original_width) || y + dy >= (*original_height)){
							pixel = (*original + x * (*original_height) + y);
						}
						else{
							pixel = (*original + x + dx * (*original_height) + (y + dy));
						}

						// Aplicar peso
						blurpixel = blurpixel + pixel * pixelweight;
					}
				}

				// Escribir pixel para difuminar la imagen
				(*imgblur + x * (*original_height) + y) = (blurpixel / 159);
			}
		}

		printf("gaussianKernel\n");

		original = imgblur;
	}
}

__global__
void sobelFilter(int *original, int *original_width, int *original_height, int *size){
	int id = threadIdx.x + blockDim.x * blockIdx.x;

	// Declaraciones
	int pix[3];
	int gradx, grady;
	int graddir, grad;

	if(id < (*size)){
		// Conseguir pixeles y calcular el gradiente y su dirección
		for (int x = 1; x <= (*original_width) - 1; x++){
			for (int y = 1; y <= (*original_height) - 1; y++){
				// Conseguir los pixeles de origen para calcular la dirección e intensidad
				pix[0] = (*original + x * (*original_height) + y);	 // pixel principal
				pix[1] = (*original + (x - 1) * (*original_height) + y); // pixel izquierdo
				pix[2] = (*original + x * (*original_height) + (y - 1)); // pixel encima

				// Conseguir valor para gradiente x
				gradx = pix[0] - pix[1];

				// Conseguir valor para gradiente y
				grady = pix[0] - pix[2];

				// Calcular dirección del gradiente
				// Queremos redondearlo a 0, 1, 2, 3 que representa 0, 45, 90, 135 grados
				graddir = (int)(abs(atan2f(grady, gradx)) + 0.22) * 80;

				// Guardar dirección del gradiente
				(*imggraddir + x * (*original_height) + y) = graddir;

				// Calcular gradiente
				grad = (int)sqrtf(gradx * gradx + grady * grady) * 2;

				// Guardar pixel
				(*imggrad + x * (*original_height) + y) = grad;
			}
		}

		for(int x = 0; x < (*original_width); x++){
			(*imggrad + x * (*original_height) + 0) = 0;
			(*imggrad + x * (*original_height) + 1) = 0;
			(*imggrad + x * (*original_height) + 2) = 0;
			(*imggrad + x * (*original_height) + ((*original_height) - 1)) = 0;
		}

		for(int y = 0; y < (*original_height); y++){
			(*imggrad + 0 * (*original_height) + y) = 0;
			(*imggrad + 1 * (*original_height) + y) = 0;
			(*imggrad + 2 * (*original_height) + y) = 0;
			(*imggrad + ((*original_width) - 1) * (*original_height) + y) = 0;
			(*imggrad + ((*original_width) - 2) * (*original_height) + y) = 0;
			(*imggrad + ((*original_width) - 3) * (*original_height) + y) = 0;
		}
		
		printf("sobelFilter\n");
		
		original = imggrad;
	}
}

void edgeDetection(int *image_pointer, int width, int height){
	// <<< Número de bloques, número de hebras >>>
	dim3 unBloque(64,1,1);
	dim3 bloques((width/64)+1, 1, 1);
	int *img_size = (int *)malloc(sizeof(int));
	int *img_width = (int *)malloc(sizeof(int));
	int *img_height = (int *)malloc(sizeof(int));
	int **gpu_img = NULL;
	int *gpu_img_size = NULL;
	int *gpu_width = NULL;
	int *gpu_height = NULL;

	// Declarar imagen para guardar el difuminado
	int *imgblur = NULL;

	// Declarar imagen para guardar la intensidad del gradiente
	int *imggrad = NULL;

	// Declarar imagen para guardar la dirección del gradiente
	int *imggraddir = NULL;

	*img_size = width * height;
	*img_width = width;
	*img_height = height;

	// Reserva de memoria en la GPU
	hipMalloc((void **) gpu_img, (*img_size)*sizeof(int));
	hipMalloc((void **) gpu_img_size, sizeof(int));
	hipMalloc((void **) gpu_width, sizeof(int));
	hipMalloc((void **) gpu_height, sizeof(int));
	hipMalloc((void **) imgblur, sizeof(int) * height * width);
	hipMalloc((void **) imggrad, sizeof(int) * height * width);
	hipMalloc((void **) imggraddir, sizeof(int) * height * width);

	// Copia de memoria en la GPU
	hipMemcpy(gpu_img, image_pointer, (*img_size)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_img_size, img_size, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_width, img_width, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_height, img_height, sizeof(int), hipMemcpyHostToDevice);

	printf("edgeDetection\n");

	// Llamada a los kernel
	gaussianKernel<<< bloques, unBloque >>>(gpu_img, gpu_width, gpu_height, gpu_img_size);
	hipDeviceSynchronize();
	sobelFilter<<< bloques, unBloque >>>(gpu_img, gpu_width, gpu_height, gpu_img_size);
	hipDeviceSynchronize();

	hipMemcpy(image_pointer, gpu_img, (*img_size)*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(gpu_img_size);
	hipFree(gpu_img);
	hipFree(gpu_width);
	hipFree(gpu_height);
	free(img_size);
	free(img_width);
	free(img_height);

	return image_pointer;
}
