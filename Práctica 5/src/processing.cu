#include "hip/hip_runtime.h"
#include <processing.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

__global__
void gaussianKernel(int *original, int width, int height, int *imgblur){
	int id = threadIdx.x + blockDim.x * blockIdx.x;

	// Declaraciones
	unsigned int blurpixel;
	signed int dx, dy;
	unsigned int pixelweight;
	unsigned int pixel;

	// Declarar el array de pesos para el difuminado gaussiano
	int weighting[5][5] = {
		{2, 4, 5, 4, 2},
		{4, 9, 12, 9, 4},
		{5, 12, 15, 12, 5},
		{4, 9, 12, 9, 4},
		{2, 4, 5, 4, 2}
	};

	if(id < width - 2){
		// Aplicar el flitro a cada pixel
		for (int y = 2; y <= height - 2; y++){
			
			// Limpiar blurpixel
			blurpixel = 0;

			// +-2 para cada pixel y calcular el peso
			for (dx = -2; dx <= 2; dx++){
				for (dy = -2; dy <= 2; dy++){
					pixelweight = weighting[dx + 2][dy + 2];

					// Conseguir pixel
					if(id + dx >= width || y + dy >= height){
						pixel = *(original + id * height + y);
					}
					else{
						pixel = *(original + (id + dx) * height + (y + dy));
					}

					// Aplicar peso
					blurpixel = blurpixel + pixel * pixelweight;
				}
			}

			// Escribir pixel para difuminar la imagen
			*(imgblur + id * height + y) = (blurpixel / 159);
		}
	}
}

__global__
void sobelFilter(int *original, int width, int height, int *imggrad, int *imggraddir){
	int id = threadIdx.x + blockDim.x * blockIdx.x;

	// Declaraciones
	int pix[3];
	int gradx, grady;
	int graddir, grad;

	if(id < width - 1){
		// Conseguir pixeles y calcular el gradiente y su dirección
		for (int y = 1; y <= height - 1; y++){
			// Conseguir los pixeles de origen para calcular la dirección e intensidad
			pix[0] = *(original + id * height + y);	 // pixel principal
			pix[1] = *(original + (id - 1) * height + y); // pixel izquierdo
			pix[2] = *(original + id * height + (y - 1)); // pixel encima

			// Conseguir valor para gradiente x
			gradx = pix[0] - pix[1];

			// Conseguir valor para gradiente y
			grady = pix[0] - pix[2];

			// Calcular dirección del gradiente
			// Queremos redondearlo a 0, 1, 2, 3 que representa 0, 45, 90, 135 grados
			graddir = (int)(abs(atan2f(grady, gradx)) + 0.22) * 80;

			// Guardar dirección del gradiente
			*(imggraddir + id * height + y) = graddir;

			// Calcular gradiente
			grad = (int)sqrtf(gradx * gradx + grady * grady) * 2;

			// Guardar pixel
			*(imggrad + id * height + y) = grad;
		}

		*(imggrad + id * height + 0) = 0;
		*(imggrad + id * height + 1) = 0;
		*(imggrad + id * height + 2) = 0;
		*(imggrad + id * height + (height - 1)) = 0;

		for(int y = 0; y < height; y++){
			*(imggrad + 0 * height + y) = 0;
			*(imggrad + 1 * height + y) = 0;
			*(imggrad + 2 * height + y) = 0;
			*(imggrad + (width - 1) * height + y) = 0;
			*(imggrad + (width - 2) * height + y) = 0;
			*(imggrad + (width - 3) * height + y) = 0;
		}
	}
}

void edgeDetection(int *image_pointer, int width, int height){
	// <<< Número de bloques, número de hebras >>>
	dim3 unBloque(64, 1, 1);
	dim3 bloques((width / 64) + 1, 1, 1);
	int *gpu_img = NULL;

	// Declarar imagen para guardar el difuminado
	int *imgblur = NULL;

	// Declarar imagen para guardar la intensidad del gradiente
	int *imggrad = NULL;

	// Declarar imagen para guardar la dirección del gradiente
	int *imggraddir = NULL;

	// Reserva de memoria en la GPU
	hipMalloc((void **) &gpu_img, sizeof(int) * (width * height));
	hipMalloc((void **) &imgblur, sizeof(int) * (width * height));
	hipMalloc((void **) &imggrad, sizeof(int) * (width * height));
	hipMalloc((void **) &imggraddir, sizeof(int) * (width * height));

	// Copia de memoria en la GPU
	hipMemcpy(gpu_img, image_pointer, sizeof(int) * (width * height), hipMemcpyHostToDevice);

	// Llamada a los kernel
	gaussianKernel<<< bloques, unBloque >>>(gpu_img, width, height, imgblur);
	hipDeviceSynchronize();
	sobelFilter<<< bloques, unBloque >>>(imgblur, width, height, imggrad, imggraddir);
	hipDeviceSynchronize();

	hipMemcpy(image_pointer, imggrad, sizeof(int) * (width * height), hipMemcpyDeviceToHost);

	hipFree(gpu_img);
	hipFree(imgblur);
	hipFree(imggrad);
	hipFree(imggraddir);
}
