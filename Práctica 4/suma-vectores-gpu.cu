
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <ctime>
#include <math.h>

using namespace std;

void readfile(vector<float> &vec, string filename){
	ifstream file;

	file.open(filename);

	if(!file.is_open()){
		cerr << "No se ha podido abrir el archivo " << filename << endl;

		exit(-1);
	}

	for(string line; getline(file, line);){
		vec.push_back(stof(line));
	}

	vec.erase(vec.begin());

	file.close();
}

// Kernel function to add the elements of two arrays
__global__
void add(float *x, float *y, float *result, int size){
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if(i < size){
		result[i] = x[i] + y[i];
	}
}

int main(void){
	clock_t begin, end;
	vector<float> vec;
	float *memoria_x, *memoria_y;
	float *gpu_x, *gpu_y;
	float *memoria_result, *gpu_result;

	memoria_x = NULL;
	memoria_y = NULL;
	gpu_x = NULL;
	gpu_y = NULL;
	memoria_x = NULL;
	gpu_y = NULL;

	readfile(vec, "data/9/input0.raw");

	// Reservar memoria para el primer array
	hipMallocManaged(&memoria_x, vec.size()*sizeof(float));
	hipMallocManaged(&gpu_x, vec.size()*sizeof(float));

	for(int i = 0; i < vec.size(); i++){
		memoria_x[i] = vec[i];
	}

	readfile(vec, "data/9/input1.raw");

	// Reservar memoria para el segundo array
	hipMallocManaged(&memoria_y, vec.size()*sizeof(float));
	hipMallocManaged(&gpu_y, vec.size()*sizeof(float));

	for(int i = 0; i < vec.size(); i++){
		memoria_y[i] = vec[i];
	}

	// Reservar memoria para el array resultante
	hipMallocManaged(&memoria_result, vec.size()*sizeof(float));
	hipMallocManaged(&gpu_result, vec.size()*sizeof(float));

	// Copiar los datos en la GPU
	hipMemcpy(gpu_x, memoria_x, sizeof(float)*vec.size(), hipMemcpyHostToDevice);
	hipMemcpy(gpu_y, memoria_y, sizeof(float)*vec.size(), hipMemcpyHostToDevice);
	hipMemcpy(gpu_result, memoria_result, sizeof(float)*vec.size(), hipMemcpyHostToDevice);

	begin = clock();

	// Llamar al kernel
	// <<< Número de bloques, número de hebras >>>
	add<<< (int)(vec.size()/256)+1, (int)vec.size() >>>(gpu_x, gpu_y, gpu_result, vec.size());

	// Esperar a que la GPU termine
	hipDeviceSynchronize();

	// Copiar los resultados en memoria
	hipMemcpy(memoria_result, gpu_result, sizeof(float)*vec.size(), hipMemcpyDeviceToHost);

	end = clock();

	for(int i = 0; i < vec.size(); i++){
		cout << memoria_result[i] << endl;
	}

	cout << "Tiempo: " << double(end - begin) / CLOCKS_PER_SEC << " segundos" << endl;

	// Free memory
	hipFree(memoria_x);
	hipFree(memoria_y);
	hipFree(memoria_result);
	hipFree(gpu_x);
	hipFree(gpu_y);
	hipFree(gpu_result);

	return 0;
}