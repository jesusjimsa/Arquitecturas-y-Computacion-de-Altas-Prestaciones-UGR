#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <ctime>
#include <math.h>

using namespace std;

void properties(){
	hipDeviceProp_t dev;
	int dev_cnt = 0;
	hipGetDeviceCount(&dev_cnt);

	cout << dev_cnt << " dispositivos" << endl;

	for(int i = 0; i < dev_cnt; i++){
		hipGetDeviceProperties(&dev, i);

		cout << "Device: " << i << endl;
		cout << "name:" << dev.name << endl;
		cout << "Compute capability " << dev.major << "." << dev.minor << endl;
		cout << "total global memory(KB): " << dev.totalGlobalMem/1024 << endl;
		cout << "shared mem per block: " << dev.sharedMemPerBlock << endl;
		cout << "regs per block: " << dev.regsPerBlock << endl;
		cout << "warp size: " << dev.warpSize << endl;
		cout << "max threads per block: " << dev.maxThreadsPerBlock << endl;
		cout << "max thread dim z:" << dev.maxThreadsDim[0] << " y:" << dev.maxThreadsDim[1] << " x:" << dev.maxThreadsDim[2] << endl;
		cout << "max grid size z:" << dev.maxGridSize[0] << " y:" << dev.maxGridSize[1] << " x:" << dev.maxGridSize[2] << endl;
		cout << "clock rate(KHz):" << dev.clockRate << endl;
		cout << "total constant memory (bytes): " << dev.totalConstMem << endl;
		cout << "multiprocessor count " << dev.multiProcessorCount << endl;
		cout << "integrated: " << dev.integrated << endl;
		cout << "async engine count: " << dev.asyncEngineCount << endl;
		cout << "memory bus width: " << dev.memoryBusWidth << endl;
		cout << "memory clock rate (KHz): " << dev.memoryClockRate << endl;
		cout << "L2 cache size (bytes): " << dev.l2CacheSize << endl;
		cout << "max threads per SM: " << dev.maxThreadsPerMultiProcessor << endl;
	}
}

void readfile(vector<float> &vec, string filename){
	ifstream file;

	file.open(filename);

	if(!file.is_open()){
		cerr << "No se ha podido abrir el archivo " << filename << endl;

		exit(-1);
	}

	for(string line; getline(file, line);){
		vec.push_back(stof(line));
	}

	vec.erase(vec.begin());

	file.close();
}

// Kernel function to add the elements of two arrays
__global__
void add(float *x, float *y, float *result, int size){
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if(i < size){
		result[i] = x[i] + y[i];
	}
}

int main(void){
	clock_t begin, end;
	vector<float> vec;
	float *memoria_x, *memoria_y;
	float *gpu_x, *gpu_y;
	float *memoria_result, *gpu_result;
	ofstream file;


	memoria_x = NULL;
	memoria_y = NULL;
	gpu_x = NULL;
	gpu_y = NULL;
	memoria_x = NULL;
	gpu_y = NULL;

	// Imprimir características
	properties();

	readfile(vec, "data/9/input0.raw");

	// Reservar memoria para el primer array
	hipMallocManaged(&memoria_x, vec.size()*sizeof(float));
	hipMallocManaged(&gpu_x, vec.size()*sizeof(float));

	for(int i = 0; i < vec.size(); i++){
		memoria_x[i] = vec[i];
	}

	readfile(vec, "data/9/input1.raw");

	// Reservar memoria para el segundo array
	hipMallocManaged(&memoria_y, vec.size()*sizeof(float));
	hipMallocManaged(&gpu_y, vec.size()*sizeof(float));

	for(int i = 0; i < vec.size(); i++){
		memoria_y[i] = vec[i];
	}

	// Reservar memoria para el array resultante
	hipMallocManaged(&memoria_result, vec.size()*sizeof(float));
	hipMallocManaged(&gpu_result, vec.size()*sizeof(float));

	// Copiar los datos en la GPU
	hipMemcpy(gpu_x, memoria_x, sizeof(float)*vec.size(), hipMemcpyHostToDevice);
	hipMemcpy(gpu_y, memoria_y, sizeof(float)*vec.size(), hipMemcpyHostToDevice);
	hipMemcpy(gpu_result, memoria_result, sizeof(float)*vec.size(), hipMemcpyHostToDevice);

	begin = clock();

	// Llamar al kernel
	// <<< Número de bloques, número de hebras >>>
	add<<< (int)(vec.size()/256)+1, (int)vec.size() >>>(gpu_x, gpu_y, gpu_result, vec.size());

	// Esperar a que la GPU termine
	hipDeviceSynchronize();

	// Copiar los resultados en memoria
	hipMemcpy(memoria_result, gpu_result, sizeof(float)*vec.size(), hipMemcpyDeviceToHost);

	end = clock();

	file.open("result.raw");

	for(int i = 0; i < vec.size(); i++){
		file << memoria_result[i] << endl;
	}

	file.close();

	cout << "Tiempo: " << double(end - begin) / CLOCKS_PER_SEC << " segundos" << endl;

	// Free memory
	hipFree(memoria_x);
	hipFree(memoria_y);
	hipFree(memoria_result);
	hipFree(gpu_x);
	hipFree(gpu_y);
	hipFree(gpu_result);

	return 0;
}